#include "hip/hip_runtime.h"
/*
 * For licensing see accompanying LICENSE file.
 * Copyright (C) 2023 Apple Inc. All Rights Reserved.
 */

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/cuda/HIPContext.h>
#include <ATen/ATen.h>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <ATen/AccumulateType.h>
#include <c10/cuda/CUDAException.h>

#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm_batched.h>
#include <cutlass/gemm/gemm.h>
#include <cutlass/layout/matrix.h>
#include <cutlass/numeric_types.h>
#include <cutlass/array.h>
#include <cutlass/epilogue/thread/linear_combination.h>
#include <cutlass/gemm/threadblock/threadblock_swizzle.h>

#include <hip/hip_fp16.h>

#include <mma.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#define CUDA_NUM_THREADS 1024


#define CUDA_CHECK(status) \
    do { \
        hipError_t err = status; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            throw std::runtime_error(hipGetErrorString(err)); \
        } \
    } while (0)

template <typename scalar_t>
__global__ void clusten_av_cuda_forward_kernel_opt(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> attn,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> v,
    const torch::PackedTensorAccessor32<int64_t,3,torch::DefaultPtrTraits> nbhd_idx,
    torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> feat,
    const int length,
    const int batch_size,
    const int heads,
    const int nbhd_size,
    const int dim) {

    const int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (z >= batch_size * heads) return;

    const int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= length) return;

    const int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (c >= dim) return;

    const int b = z / heads;
    const int h = z % heads;

    scalar_t updt = scalar_t(0);
    for (int ni = 0; ni < nbhd_size; ++ni) {
            const int64_t nbi = nbhd_idx[b][i][ni];
            updt += attn[b][h][i][ni] * v[b][h][nbi][c];
    }
    feat[b][h][i][c] = updt;
}

torch::Tensor clusten_av_cuda_forward_opt(
    const torch::Tensor &attn,
    const torch::Tensor &v,
    const torch::Tensor &nbhd_idx) {

    const int64_t batch_size = attn.size(0);
    const int64_t heads = attn.size(1);
    const int64_t length = attn.size(2);
    const int64_t dim = v.size(3);
    const int64_t nbhd_size = nbhd_idx.size(2);
    const int zsize = batch_size * heads;

    auto feat = torch::zeros({batch_size, heads, length, dim}, v.options());

    const int CHANNELTHREADS = std::min<int64_t>(32, dim);
    const int TOKENTHREADS = std::min<int64_t>(8, length);
    const int BATCHTHREADS = std::max(1, CUDA_NUM_THREADS / (TOKENTHREADS * CHANNELTHREADS));

    const dim3 blocks(
        (dim + CHANNELTHREADS - 1) / CHANNELTHREADS,
        (length + TOKENTHREADS - 1) / TOKENTHREADS,
        (zsize + BATCHTHREADS - 1) / BATCHTHREADS);
    const dim3 threads(CHANNELTHREADS, TOKENTHREADS, BATCHTHREADS);

    const auto stream = c10::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        attn.scalar_type(),
        "clusten_av_cuda_forward",
        ([&] {
            clusten_av_cuda_forward_kernel_opt<scalar_t><<<blocks, threads, 0, stream>>>(
                attn.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>(),
                v.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>(),
                nbhd_idx.packed_accessor32<int64_t,3,torch::DefaultPtrTraits>(),
                feat.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>(),
                length, batch_size, heads, nbhd_size, dim);
        })
    );

    return feat;
}

template <typename scalar_t>
__global__ void clusten_av_cuda_backward_kernel_opt(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_feat,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> attn,
    const torch::PackedTensorAccessor32<int64_t,3,torch::DefaultPtrTraits> nbhd_idx,
    torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_v,
    const int length,
    const int batch_size,
    const int heads,
    const int nbhd_size,
    const int dim,
    const size_t d_v_numel) {

    const int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (z < batch_size * heads) {
            const int i = blockIdx.y * blockDim.y + threadIdx.y;
            if (i < length) {
                    const int c = blockIdx.x * blockDim.x + threadIdx.x;
                    if (c < dim) {
                            const int b = z / heads;
                            const int h = z - b * heads;
                            int64_t nbi;
                            size_t index;
                            #pragma unroll
                            for (unsigned int ni=0; ni < nbhd_size; ++ni) {
                                    nbi = nbhd_idx[b][i][ni];
                                    index = b*d_v.stride(0) + h*d_v.stride(1) + nbi*d_v.stride(2) + c;
                                    at::native::fastAtomicAdd(d_v.data(), index, d_v_numel, d_feat[b][h][i][c] * attn[b][h][i][ni], true);
                            }
                    }
            }
    }
}

template <typename scalar_t, int ELEMENTS_PER_THREAD>
__global__ void clusten_av_attn_cuda_backward_kernel_warp_impl(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_feat,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> v,
    const torch::PackedTensorAccessor32<int64_t,3,torch::DefaultPtrTraits> nbhd_idx,
    torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_attn,
    const int length,
    const int batch_size,
    const int heads,
    const int nbhd_size,
    const int dim) {

    const int b = blockIdx.z;
    const int h = blockIdx.y;
    const int l = blockIdx.x;
    const int tid = threadIdx.x;

    scalar_t local_dfeat[ELEMENTS_PER_THREAD];
    #pragma unroll
    for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
            int c = tid + i * 32;
            if (c < dim) {
                    local_dfeat[i] = d_feat[b][h][l][c];
            } else {
                    local_dfeat[i] = scalar_t(0);
            }
    }

    for (int ni = 0; ni < nbhd_size; ni++) {
            int64_t nbi = nbhd_idx[b][l][ni];
            scalar_t sum = scalar_t(0);
            #pragma unroll
            for (int i = 0; i < ELEMENTS_PER_THREAD; i++) {
                    int c = tid + i * 32;
                    if (c < dim) {
                            sum += local_dfeat[i] * v[b][h][nbi][c];
                    }
            }

            cg::thread_block_tile<32> tile = cg::tiled_partition<32>(cg::this_thread_block());
            scalar_t total = cg::reduce(tile, sum, cg::plus<scalar_t>());

            if (tile.thread_rank() == 0) {
                    d_attn[b][h][l][ni] = total;
            }
    }
}

template <typename scalar_t>
void launch_clusten_av_attn_cuda_backward_kernel_warp(
    int dim,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_feat,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> v,
    const torch::PackedTensorAccessor32<int64_t,3,torch::DefaultPtrTraits> nbhd_idx,
    torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_attn,
    int length, int batch_size, int heads, int nbhd_size,
    hipStream_t stream) {

    const int elements_per_thread = (dim + 31) / 32;
    dim3 blocks(length, heads, batch_size);
    dim3 threads(32);

    // Select template instantiation based on elements_per_thread
    switch (elements_per_thread) {
        case 1:
            clusten_av_attn_cuda_backward_kernel_warp_impl<scalar_t, 1><<<blocks, threads, 0, stream>>>(
                d_feat, v, nbhd_idx, d_attn, length, batch_size, heads, nbhd_size, dim);
            break;
        case 2:
            clusten_av_attn_cuda_backward_kernel_warp_impl<scalar_t, 2><<<blocks, threads, 0, stream>>>(
                d_feat, v, nbhd_idx, d_attn, length, batch_size, heads, nbhd_size, dim);
            break;
        case 3:
            clusten_av_attn_cuda_backward_kernel_warp_impl<scalar_t, 3><<<blocks, threads, 0, stream>>>(
                d_feat, v, nbhd_idx, d_attn, length, batch_size, heads, nbhd_size, dim);
            break;
        case 4:
            clusten_av_attn_cuda_backward_kernel_warp_impl<scalar_t, 4><<<blocks, threads, 0, stream>>>(
                d_feat, v, nbhd_idx, d_attn, length, batch_size, heads, nbhd_size, dim);
            break;
        // Add more cases if needed for larger dimensions
        default:
            AT_ERROR("Unsupported elements_per_thread: ", elements_per_thread);
    }
}

std::vector<torch::Tensor> clusten_av_cuda_backward_opt(
    const torch::Tensor &d_feat,
    const torch::Tensor &attn,
    const torch::Tensor &v,
    const torch::Tensor &nbhd_idx) {

    int64_t batch_size = attn.size(0);
    int64_t heads = attn.size(1);
    int64_t length = attn.size(2);
    int64_t dim = v.size(3);
    int64_t nbhd_size = nbhd_idx.size(2);
    int zsize = batch_size * heads;

    auto d_attn = torch::zeros_like(attn);
    auto d_v = torch::zeros_like(v);
    hipStream_t stream = c10::cuda::getCurrentCUDAStream();

    hipStream_t stream1, stream2;
    CUDA_CHECK(hipStreamCreate(&stream1));
    CUDA_CHECK(hipStreamCreate(&stream2));

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(attn.scalar_type(), "clusten_av_cuda_backward", ([&] {
        const auto d_feat_a = d_feat.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();
        const auto attn_a = attn.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();
        const auto v_a = v.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();
        const auto nbhd_idx_a = nbhd_idx.packed_accessor32<int64_t,3,torch::DefaultPtrTraits>();
        auto d_attn_a = d_attn.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();
        auto d_v_a = d_v.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();

        const size_t d_v_numel = d_v.numel();

        // Launch d_attn kernel in stream1
        launch_clusten_av_attn_cuda_backward_kernel_warp<scalar_t>(
                                    dim, d_feat_a, v_a, nbhd_idx_a, d_attn_a,
                                    length, batch_size, heads, nbhd_size, stream1);

        // Launch d_v kernel in stream2
        int CHANNELTHREADS = min(int64_t(CUDA_NUM_THREADS), dim);
        int TOKENTHREADS = min(int64_t(CUDA_NUM_THREADS / CHANNELTHREADS), length);
        int BATCHTHREADS = max(1, CUDA_NUM_THREADS / (TOKENTHREADS * CHANNELTHREADS));
        const dim3 blocks_v(
                            (dim + CHANNELTHREADS - 1) / CHANNELTHREADS,
                            (length + TOKENTHREADS - 1) / TOKENTHREADS,
                            (zsize + BATCHTHREADS - 1) / BATCHTHREADS);
        const dim3 threads_v(CHANNELTHREADS, TOKENTHREADS, BATCHTHREADS);

        clusten_av_cuda_backward_kernel_opt<scalar_t><<<blocks_v, threads_v, 0, stream2>>>(
            d_feat_a, attn_a, nbhd_idx_a, d_v_a,
            length, batch_size, heads, nbhd_size, dim, d_v_numel);
    }));

    CUDA_CHECK(hipStreamSynchronize(stream1));
    CUDA_CHECK(hipStreamSynchronize(stream2));
    CUDA_CHECK(hipStreamDestroy(stream1));
    CUDA_CHECK(hipStreamDestroy(stream2));

    return {d_attn, d_v.to(v.dtype())};
}


template <typename scalar_t>
__global__ void clusten_av_cuda_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> attn,               // b x h x n x m
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> v,                  // b x h x n x c
    const torch::PackedTensorAccessor32<int64_t,3,torch::DefaultPtrTraits> nbhd_idx,            // b x n x m
    torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> feat,                     // b x n x c
    const int length,               // n
    const int batch_size,           // b
    const int heads,                // h
    const int nbhd_size,            // m
    const int dim) {                // c

    const int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (z < batch_size * heads){
        const int i = blockIdx.y * blockDim.y + threadIdx.y;
        if (i < length){
            const int c = blockIdx.x * blockDim.x + threadIdx.x;
            if (c < dim){
                const int b = z / heads;
                const int h = z - b * heads;
                int64_t nbi;
                // calculate a@v
                scalar_t updt = scalar_t(0);
                #pragma unroll
                for (unsigned int ni=0; ni < nbhd_size; ++ni) {
                    nbi = nbhd_idx[b][i][ni];
                    updt += attn[b][h][i][ni] * v[b][h][nbi][c];
                }
                feat[b][h][i][c] = updt;
            }
        }
    }
}


torch::Tensor clusten_av_cuda_forward(
    const torch::Tensor &attn,             
    const torch::Tensor &v,               
    const torch::Tensor &nbhd_idx) { 

    int64_t batch_size = attn.size(0);
    int64_t heads = attn.size(1);
    int64_t length = attn.size(2);
    int64_t dim = v.size(3);
    int64_t nbhd_size = nbhd_idx.size(2);
    int zsize = batch_size * heads;

    int CHANNELTHREADS = min(int64_t(CUDA_NUM_THREADS), dim);
    int TOKENTHREADS = min(int64_t(CUDA_NUM_THREADS / CHANNELTHREADS), length);
    int BATCHTHREADS = max(1, CUDA_NUM_THREADS / (TOKENTHREADS * CHANNELTHREADS));

    auto feat = torch::zeros(
            {batch_size, heads, length, dim}, v.options());

    const auto stream = c10::cuda::getCurrentCUDAStream();
    const dim3 blocks(
            (dim + CHANNELTHREADS - 1) / CHANNELTHREADS,
            (length + TOKENTHREADS - 1) / TOKENTHREADS,
            (zsize + BATCHTHREADS - 1) / BATCHTHREADS);
    const dim3 threads(CHANNELTHREADS, TOKENTHREADS, BATCHTHREADS);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(attn.scalar_type(), "clusten_av_cuda_forward", ([&] {
        const auto attn_a = attn.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();
        const auto v_a = v.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();
        const auto nbhd_idx_a = nbhd_idx.packed_accessor32<int64_t,3,torch::DefaultPtrTraits>();
        auto feat_a = feat.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();

        clusten_av_cuda_forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
                attn_a, v_a, nbhd_idx_a, feat_a,
                length, batch_size, heads, nbhd_size, dim);
    }));
    return feat;
}


template <typename scalar_t>
__global__ void clusten_av_cuda_backward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_feat,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> attn,
    const torch::PackedTensorAccessor32<int64_t,3,torch::DefaultPtrTraits> nbhd_idx,
    torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_v,
    const int length,
    const int batch_size,
    const int heads,
    const int nbhd_size,
    const int dim,
    const size_t d_v_numel) {

    const int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (z < batch_size * heads){
        const int i = blockIdx.y * blockDim.y + threadIdx.y;
        if (i < length){
            const int c = blockIdx.x * blockDim.x + threadIdx.x;
            if (c < dim){
                const int b = z / heads;
                const int h = z - b * heads;
                int64_t nbi;
                size_t index;
                #pragma unroll
                for (unsigned int ni=0; ni < nbhd_size; ++ni) {
                    nbi = nbhd_idx[b][i][ni];
                    // calculate d_v = att * d_feat
                    index = b*d_v.stride(0) + h*d_v.stride(1) + nbi*d_v.stride(2) + c;
                    at::native::fastAtomicAdd(d_v.data(), index, d_v_numel, d_feat[b][h][i][c] * attn[b][h][i][ni], true);
                    // atomicAdd(&(d_v[b][h][nbi][c]), d_feat[b][h][i][c] * attn[b][h][i][ni]); // avoid race condition
                }
            }
        }
    }
}

template <typename scalar_t>
__global__ void clusten_av_attn_cuda_backward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_feat,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> v,
    const torch::PackedTensorAccessor32<int64_t,3,torch::DefaultPtrTraits> nbhd_idx,
    torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_attn,
    const int length,
    const int batch_size,
    const int heads,
    const int nbhd_size,
    const int dim) {

    const int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (z < batch_size * heads){
            const int i = blockIdx.y * blockDim.y + threadIdx.y;
            if (i < length){
                    const int ni = blockIdx.x * blockDim.x + threadIdx.x;
                    if (ni < nbhd_size){
                            const int b = z / heads;
                            const int h = z - b * heads;
                            int64_t nbi = nbhd_idx[b][i][ni];
                            scalar_t updt = scalar_t(0);
                            #pragma unroll
                            for (unsigned int c=0; c < dim; ++c) {
                                    // calculate d_attn = v * d_feat
                                    updt += v[b][h][nbi][c] * d_feat[b][h][i][c];
                            }
                            d_attn[b][h][i][ni] = updt;
                    }
            }
    }
}

std::vector<torch::Tensor> clusten_av_cuda_backward(
    const torch::Tensor &d_feat,
    const torch::Tensor &attn,
    const torch::Tensor &v,
    const torch::Tensor &nbhd_idx) {

    int64_t batch_size = attn.size(0);
    int64_t heads = attn.size(1);
    int64_t length = attn.size(2);
    int64_t dim = v.size(3);
    int64_t nbhd_size = nbhd_idx.size(2);
    int zsize = batch_size * heads;

    int CHANNELTHREADS = min(int64_t(CUDA_NUM_THREADS), dim);
    int TOKENTHREADS = min(int64_t(CUDA_NUM_THREADS / CHANNELTHREADS), length);
    int BATCHTHREADS = max(1, CUDA_NUM_THREADS / (TOKENTHREADS* CHANNELTHREADS));

    int NBHDTHREADS = min(int64_t(CUDA_NUM_THREADS), nbhd_size);
    int TOKENTHREADS_NB = min(int64_t(CUDA_NUM_THREADS / NBHDTHREADS), length);
    int BATCHTHREADS_NB = max(1, CUDA_NUM_THREADS / (TOKENTHREADS_NB* NBHDTHREADS));

    auto d_attn = torch::zeros_like(attn);
    auto d_v = torch::zeros_like(v);

    const auto stream = c10::cuda::getCurrentCUDAStream();

    const dim3 blocks(
            (dim + CHANNELTHREADS - 1) / CHANNELTHREADS,
            (length + TOKENTHREADS - 1) / TOKENTHREADS,
            (zsize + BATCHTHREADS - 1) / BATCHTHREADS);
    const dim3 threads(CHANNELTHREADS, TOKENTHREADS, BATCHTHREADS);

    const dim3 blocks_nb(
            (nbhd_size + NBHDTHREADS - 1) / NBHDTHREADS,
            (length + TOKENTHREADS_NB - 1) / TOKENTHREADS_NB,
            (zsize + BATCHTHREADS_NB - 1) / BATCHTHREADS_NB);
    const dim3 threads_nb(NBHDTHREADS, TOKENTHREADS_NB, BATCHTHREADS_NB);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(attn.scalar_type(), "clusten_av_cuda_backward", ([&] {
        const auto d_feat_a = d_feat.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();
        const auto attn_a = attn.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();
        const auto v_a = v.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();
        const auto nbhd_idx_a = nbhd_idx.packed_accessor32<int64_t,3,torch::DefaultPtrTraits>();
        auto d_attn_a = d_attn.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();
        auto d_v_a = d_v.packed_accessor32<scalar_t,4,torch::DefaultPtrTraits>();

        const size_t d_v_numel = d_v.numel();
        clusten_av_cuda_backward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
                d_feat_a, attn_a, nbhd_idx_a, d_v_a,
                length, batch_size, heads, nbhd_size, dim, d_v_numel);
        clusten_av_attn_cuda_backward_kernel<scalar_t><<<blocks_nb, threads_nb, 0, stream>>>(
                d_feat_a, v_a, nbhd_idx_a, d_attn_a,
                length, batch_size, heads, nbhd_size, dim);
    }));

    return {d_attn, d_v.to(v.dtype())};
}